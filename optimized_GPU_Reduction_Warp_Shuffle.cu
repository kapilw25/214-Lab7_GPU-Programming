#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cstdlib> // for std::atoi

const int N = 131072;  // Define N here

__global__ void optimizedReductionWarpShuffle(int* input, int* output, int size) {
    extern __shared__ int sharedData[];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        sharedData[threadIdx.x] = input[tid];
        __syncthreads();

        // Loop unrolling for the last warp
        for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
            if (threadIdx.x < s) {
                sharedData[threadIdx.x] += sharedData[threadIdx.x + s];
            }
            __syncthreads();
        }

        if (threadIdx.x < 32) {
            // Warp shuffle for final warp reduction
            for (int offset = warpSize / 2; offset > 0; offset /= 2) {
                sharedData[threadIdx.x] += __shfl_down_sync(0xffffffff, sharedData[threadIdx.x], offset);
            }
        }

        if (threadIdx.x == 0) {
            output[blockIdx.x] = sharedData[0];
        }
    }
}


int main(int argc, char *argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <grid_size> <block_size>" << std::endl;
        return 1;
    }

    int gridSize = std::atoi(argv[1]);
    int blockSize = std::atoi(argv[2]);

    int* data;
    int* results;
    hipMallocManaged(&data, N * sizeof(int));
    hipMallocManaged(&results, gridSize * sizeof(int));

    for (int i = 0; i < N; i++) {
        data[i] = i;
    }

    auto start = std::chrono::high_resolution_clock::now();
    optimizedReductionWarpShuffle<<<gridSize, blockSize, blockSize * sizeof(int)>>>(data, results, N);
    hipDeviceSynchronize();
    int sum = 0;
    for (int i = 0; i < gridSize; i++) {
        sum += results[i];
    }
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed = end - start;
    std::cout << "optimized_GPU_Reduction_Warp_Shuffle Sum: " << sum << " Time: " << elapsed.count() << " seconds" << std::endl;

    hipFree(data);
    hipFree(results);

    return 0;
}
