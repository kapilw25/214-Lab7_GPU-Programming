#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cstdlib> // for std::atoi

const int N = 131072;

// Kernel function for optimized reduction with loop unrolling
__global__ void optimizedReductionLoopUnroll(int* input, int* output, int size) {
    extern __shared__ int sharedData[];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        sharedData[threadIdx.x] = input[tid];
        __syncthreads();

        // Loop unrolling for the last warp
        for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
            if (threadIdx.x < s) {
                sharedData[threadIdx.x] += sharedData[threadIdx.x + s];
            }
            __syncthreads();
        }

        if (threadIdx.x < 32) {
            // Further unroll to avoid the last few syncs
            volatile int* vsmem = sharedData;
            vsmem[threadIdx.x] += vsmem[threadIdx.x + 32];
            vsmem[threadIdx.x] += vsmem[threadIdx.x + 16];
            vsmem[threadIdx.x] += vsmem[threadIdx.x + 8];
            vsmem[threadIdx.x] += vsmem[threadIdx.x + 4];
            vsmem[threadIdx.x] += vsmem[threadIdx.x + 2];
            vsmem[threadIdx.x] += vsmem[threadIdx.x + 1];
        }

        if (threadIdx.x == 0) {
            output[blockIdx.x] = sharedData[0];
        }
    }
}


int main(int argc, char *argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <grid_size> <block_size>" << std::endl;
        return 1;
    }

    int gridSize = std::atoi(argv[1]);
    int blockSize = std::atoi(argv[2]);

    int* data;
    int* results;
    hipMallocManaged(&data, N * sizeof(int));
    hipMallocManaged(&results, gridSize * sizeof(int));

    for (int i = 0; i < N; i++) {
        data[i] = i;
    }

    auto start = std::chrono::high_resolution_clock::now();
    optimizedReductionLoopUnroll<<<gridSize, blockSize, blockSize * sizeof(int)>>>(data, results, N);
    hipDeviceSynchronize();
    int sum = 0;
    for (int i = 0; i < gridSize; i++) {
        sum += results[i];
    }
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed = end - start;
    std::cout << "optimized_GPU_Reduction_Loop_Unroll Sum: " << sum << " Time: " << elapsed.count() << " seconds" << std::endl;

    hipFree(data);
    hipFree(results);

    return 0;
}
